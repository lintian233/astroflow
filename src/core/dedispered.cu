#include "hip/hip_runtime.h"
#include "data.h"
#include "gpucal.h"
#include "marcoutils.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <hip/hip_vector_types.h>
#include "rfimarker.h"
// timeit
#include <chrono>

// cuda atomicAdd

#define CHECK_CUDA(call)                                                       \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << " at "         \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

namespace gpucal {

// Shared memory based dedispersion kernel implementing Algorithm 3 (adapted for pre-binned data)
template <typename T>
__global__ void
dedispersion_shared_memory_kernel(uint64_t *output, T *input, int *delay_table,
                                  size_t dm_steps, int time_downsample, 
                                  size_t down_ndata, size_t nchans, 
                                  size_t chan_start, size_t chan_end,
                                  size_t start, size_t shared_mem_size) {
  
  // Get thread and block indices
  size_t dmidx = blockIdx.y;
  size_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (dmidx >= dm_steps || tidx >= down_ndata) {
    return;
  }
  
  // Shared memory buffer to store local copy of x(f,t)
  extern __shared__ char shared_buffer[];
  T* Bloc = reinterpret_cast<T*>(shared_buffer);
  
  // Initialize local accumulator
  uint64_t Sloc = 0;
  
  // Initial time index (already in downsampled space)
  size_t Tini = start + blockIdx.x * blockDim.x;
  
  // Calculate number of channels per iteration
  size_t Dch = (chan_end - chan_start + 1);
  size_t Nch = min(shared_mem_size / blockDim.x, Dch);
  
  // Process channels in chunks
  for (size_t c = 0; c < Dch; c += Nch) {
    size_t channels_this_iter = min(Nch, Dch - c);
    
    // Data segment is stored into shared memory
    for (size_t ch_offset = 0; ch_offset < channels_this_iter; ++ch_offset) {
      size_t chan = chan_start + c + ch_offset;
      if (chan < chan_end && threadIdx.x < blockDim.x) {
        // Calculate the time index with dedispersion delay (in downsampled space)
        int original_delay = delay_table[dmidx * Dch + chan - chan_start];
        size_t delay_in_bins = original_delay / time_downsample; // 转换为降采样后的延迟
        size_t time_idx = Tini + threadIdx.x + delay_in_bins;
        
        // Bounds checking
        if (time_idx < down_ndata) {
          Bloc[ch_offset * blockDim.x + threadIdx.x] = 
            input[chan + time_idx * nchans];
        } else {
          Bloc[ch_offset * blockDim.x + threadIdx.x] = 0;
        }
      }
    }
    
    // Synchronize threads
    __syncthreads();
    
    // Dedisperse local data into accumulators
    for (size_t l = 0; l < channels_this_iter; ++l) {
      if (threadIdx.x < blockDim.x) {
        Sloc += Bloc[l * blockDim.x + threadIdx.x];
      }
    }
    
    // Synchronize before next iteration
    __syncthreads();
  }
  
  // Store local results into output DM(dm,t)
  if (tidx < down_ndata) {
    output[dmidx * down_ndata + tidx] = Sloc;
  }
}

template <typename T>
__global__ void
dedispersion_kernel(uint64_t *output, T *input, int *delay_table,
                    size_t dm_steps, size_t down_ndata, int time_downsample,
                    size_t nchans, size_t chan_start, size_t chan_end,
                    size_t start) {
  size_t dmidx = blockIdx.y;
  size_t down_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dmidx >= dm_steps)
    return;
    
  if (down_idx >= down_ndata)
    return;

  // 在已经降采样的数据上进行去色散
  size_t base_idx = down_idx + start;
  uint64_t sum = 0;
  for (size_t chan = chan_start; chan < chan_end; ++chan) {
    // 延迟表中的延迟已经按照原始tsamp计算，需要除以time_downsample转换为降采样后的延迟
    int original_delay = delay_table[dmidx * (chan_end - chan_start + 1) + chan - chan_start];
    size_t delay_in_bins = original_delay / time_downsample; // 参数化的time_downsample
    size_t target_idx = base_idx + delay_in_bins;
    
    if (target_idx < down_ndata) {
      sum += input[chan + target_idx * nchans];
    }
  }
  output[dmidx * down_ndata + down_idx] = sum;
}

__global__ void
pre_calculate_dedispersion_kernel(int *delay_table, float dm_low, float dm_high,
                                  float dm_step, size_t chan_start,
                                  size_t chan_end, double *freq_table,
                                  float ref_freq_value, double tsamp) {

  size_t dmidx = blockDim.x * blockIdx.x + threadIdx.x;
  float dm = dm_low + (blockDim.x * blockIdx.x + threadIdx.x) * (dm_step);
  if (dm > dm_high)
    return;
  size_t chan = blockDim.y * blockIdx.y + threadIdx.y + chan_start;
  if (chan > chan_end)
    return;

  double freq = freq_table[chan];
  float ref_2 = ref_freq_value * ref_freq_value;
  float freq_2 = freq * freq;

  float delay = 4148.808f * dm * (1.0f / freq_2 - 1.0f / ref_2);
  delay_table[dmidx * (chan_end - chan_start + 1) + chan - chan_start] =
      static_cast<int>(roundf(delay / tsamp));
}

// 时间分bin降采样kernel - 将连续的时间样本累加到bin中 (优化版本，使用1D配置)
template <typename T>
__global__ void
time_binning_kernel(T *output, T *input, size_t nchans, size_t ndata, 
                   int time_downsample, size_t down_ndata) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t total_elements = nchans * down_ndata;
  
  if (idx >= total_elements) {
    return;
  }
  
  // 计算当前元素的channel和time index
  size_t chan = idx % nchans;
  size_t down_idx = idx / nchans;
  
  // 计算当前bin的起始和结束时间索引
  size_t start_time = down_idx * time_downsample;
  size_t end_time = min(start_time + time_downsample, ndata);
  
  // 累加当前bin内的所有时间样本
  uint64_t sum = 0;
  for (size_t t = start_time; t < end_time; ++t) {
    sum += input[chan + t * nchans];
  }
  
  // 存储累加结果
  output[chan + down_idx * nchans] = static_cast<T>(sum);
}

template <typename T>
dedisperseddata_uint8 dedispered_fil_cuda(Filterbank &fil, float dm_low,
                                    float dm_high, float freq_start,
                                    float freq_end, float dm_step, int ref_freq,
                                    int time_downsample, float t_sample, 
                                    std::string mask_file, bool use_shared_memory) {

  // get all cuda devices
  int device_count;
  CHECK_CUDA(hipGetDeviceCount(&device_count));
  if (device_count == 0) {
    throw std::runtime_error("No CUDA devices found");
  }

  int device_id;
  if (device_count == 1) {
    device_id = 0;
  } else if (device_count == 4) {
    device_id = 2;
  } else {
    device_id = 0;
  }
  // print device info
  hipDeviceProp_t device_prop;
  CHECK_CUDA(hipGetDeviceProperties(&device_prop, device_id));
  printf("Using device %d: %s\n", device_id, device_prop.name);

  CHECK_CUDA(hipSetDevice(device_id));

  float fil_freq_min = fil.frequency_table[0];
  float fil_freq_max = fil.frequency_table[fil.nchans - 1];

  if (freq_start < fil_freq_min || freq_end > fil_freq_max) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Frequency range [%.3f-%.3f MHz] out of filterbank range "
             "[%.3f-%.3f MHz]",
             freq_start, freq_end, fil_freq_min, fil_freq_max);
    throw std::invalid_argument(error_msg);
  }
  
  size_t chan_start =
      static_cast<size_t>((freq_start - fil_freq_min) /
                          (fil_freq_max - fil_freq_min) * (fil.nchans - 1));
  size_t chan_end =
      static_cast<size_t>((freq_end - fil_freq_min) /
                          (fil_freq_max - fil_freq_min) * (fil.nchans - 1));

  chan_start = std::max(static_cast<size_t>(0), chan_start);
  chan_end = std::min(static_cast<size_t>(fil.nchans - 1), chan_end);

  if (chan_start >= fil.nchans || chan_end >= fil.nchans) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid channel range [%zu-%zu] for %d channels", chan_start,
             chan_end, fil.nchans);
    throw std::invalid_argument(error_msg);
  }

  const size_t nchans = fil.nchans;
  const size_t dm_steps = static_cast<size_t>((dm_high - dm_low) / dm_step) + 1;
  const float ref_freq_value = ref_freq ? fil.frequency_table[chan_end]
                                        : fil.frequency_table[chan_start];

  // Calculate the full downsampled time dimensions
  const size_t down_ndata = (fil.ndata + time_downsample - 1) / time_downsample;

  int *d_delay_table;
  CHECK_CUDA(hipMallocManaged(
      &d_delay_table, dm_steps * (chan_end - chan_start + 1) * sizeof(int)));

  double *d_freq_table;
  CHECK_CUDA(hipMallocManaged(&d_freq_table, nchans * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_freq_table, fil.frequency_table,
                        nchans * sizeof(double), hipMemcpyHostToDevice));

  dim3 block_size(64, 16);
  dim3 grid_size((dm_steps + block_size.x - 1) / block_size.x,
                 (chan_end - chan_start + 1 + block_size.y - 1) / block_size.y);

  pre_calculate_dedispersion_kernel<<<grid_size, block_size>>>(
      d_delay_table, dm_low, dm_high, dm_step, chan_start, chan_end,
      d_freq_table, ref_freq_value, fil.tsamp);

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  T *d_input;
  T *d_binned_input; // 存储分bin后的数据
  T *data = static_cast<T *>(fil.data);
  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(data, fil.nchans, fil.ndata);

  CHECK_CUDA(hipMalloc(&d_input, fil.ndata * nchans * sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_input, data, fil.ndata * nchans * sizeof(T),
                        hipMemcpyHostToDevice));

  if (time_downsample > 1) {
    // 需要进行时间分bin降采样
    CHECK_CUDA(hipMalloc(&d_binned_input, down_ndata * nchans * sizeof(T)));
    
    printf("Performing time binning: %zu -> %zu time samples (factor %d)\n", 
           fil.ndata, down_ndata, time_downsample);
    
    // 使用1D grid配置来避免grid大小限制问题
    const size_t total_elements = nchans * down_ndata;
    const int threads_per_block = 256;
    const size_t blocks_needed = (total_elements + threads_per_block - 1) / threads_per_block;
    
    printf("Binning kernel config: %zu total elements, %zu blocks, %d threads per block\n", 
           total_elements, blocks_needed, threads_per_block);
    
    auto binning_start = std::chrono::high_resolution_clock::now();
    time_binning_kernel<T><<<blocks_needed, threads_per_block>>>(
        d_binned_input, d_input, nchans, fil.ndata, time_downsample, down_ndata);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    auto binning_end = std::chrono::high_resolution_clock::now();
    auto binning_duration = std::chrono::duration_cast<std::chrono::milliseconds>(binning_end - binning_start);
    printf("Time binning completed in %lld ms\n", binning_duration.count());
    
    // 释放原始输入数据
    CHECK_CUDA(hipFree(d_input));
  } else {
    // 不需要分bin，直接使用原始数据
    printf("No time binning needed (factor = 1)\n");
    d_binned_input = d_input;
  }

  printf("Processing full data: DM steps = %zu, Time samples = %zu\n", dm_steps, down_ndata);

  uint64_t *d_output;
  CHECK_CUDA(hipMalloc(&d_output, dm_steps * down_ndata * sizeof(uint64_t)));
  CHECK_CUDA(hipMemset(d_output, 0, dm_steps * down_ndata * sizeof(uint64_t)));

  int THREADS_PER_BLOCK = 256;
  dim3 threads(THREADS_PER_BLOCK);
  dim3 grids((down_ndata + threads.x - 1) / threads.x, dm_steps);
  auto start_time = std::chrono::high_resolution_clock::now();
  if (use_shared_memory) {
    // Calculate shared memory size needed
    size_t max_shared_mem = device_prop.sharedMemPerBlock;
    size_t shared_mem_size = std::min(max_shared_mem / sizeof(T), 
                                     (chan_end - chan_start + 1) * THREADS_PER_BLOCK);
    
    // Ensure we don't exceed shared memory limits
    size_t actual_shared_mem = shared_mem_size * sizeof(T);
    
    printf("Using shared memory kernel with %zu bytes of shared memory\n", actual_shared_mem);
    dedispersion_shared_memory_kernel<T><<<grids, threads, actual_shared_mem>>>(
        d_output, d_binned_input, d_delay_table, dm_steps, time_downsample, down_ndata,
        nchans, chan_start, chan_end, 0, shared_mem_size);
  } else {
    printf("Using global memory kernel\n");
    dedispersion_kernel<T><<<grids, threads>>>(
        d_output, d_binned_input, d_delay_table, dm_steps, down_ndata, time_downsample,
        nchans, chan_start, chan_end, 0);
  }
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
  auto end_time = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

  printf("Dedispersion completed in %lld ms\n", duration.count());

  // Copy back the full result
  auto dm_array = std::shared_ptr<uint64_t[]>(
      new (std::align_val_t{4096}) uint64_t[dm_steps * down_ndata](),
      [](uint64_t *p) { operator delete[](p, std::align_val_t{4096}); });

  CHECK_CUDA(hipMemcpy(dm_array.get(), d_output,
                        dm_steps * down_ndata * sizeof(uint64_t),
                        hipMemcpyDeviceToHost));

  // 清理GPU资源
  if (time_downsample > 1) {
    CHECK_CUDA(hipFree(d_binned_input));
  } else {
    CHECK_CUDA(hipFree(d_input)); // d_binned_input == d_input when no binning
  }
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay_table));
  CHECK_CUDA(hipFree(d_freq_table));

  // Create single large dedisperseddata with all time samples
  dedisperseddata result;
  result.dm_times.emplace_back(std::move(dm_array));
  result.dm_low = dm_low;
  result.dm_high = dm_high;
  result.dm_step = dm_step;
  result.tsample = (time_downsample > 1) ? fil.tsamp * time_downsample : fil.tsamp; // 只有分bin时才更新时间分辨率
  result.filname = fil.filename;
  result.dm_ndata = dm_steps;
  result.downtsample_ndata = down_ndata;
  result.shape = {dm_steps, down_ndata};

  printf("Full dedispersion completed. Now applying preprocessing with slicing...\n");
  // 对于Filterbank，需要构造Header结构
  Header temp_header;
  temp_header.tsamp = (time_downsample > 1) ? fil.tsamp * time_downsample : fil.tsamp; // 只有分bin时才更新时间分辨率
  temp_header.filename = fil.filename;
  return preprocess_dedisperseddata_with_slicing(result, temp_header, 1, t_sample); // time_downsample设为1，因为已经分bin完成或不需要分bin
}

template <typename T>
dedisperseddata_uint8 dedisperse_spec(T *data, Header header, float dm_low,
                                float dm_high, float freq_start, float freq_end,
                                float dm_step, int ref_freq,
                                int time_downsample, float t_sample, 
                                std::string mask_file, bool use_shared_memory) { 
  // get all cuda devices
  int device_count;
  CHECK_CUDA(hipGetDeviceCount(&device_count));
  if (device_count == 0) {
    throw std::runtime_error("No CUDA devices found");
  }

  int device_id;
  if (device_count == 1) {
    device_id = 0;
  } else if (device_count == 4) {
    device_id = 2;
  } else {
    device_id = 0;
  }
  // print device info
  hipDeviceProp_t device_prop;
  CHECK_CUDA(hipGetDeviceProperties(&device_prop, device_id));
  printf("Using device %d: %s\n", device_id, device_prop.name);

  CHECK_CUDA(hipSetDevice(device_id));
  
  // Calculate frequency table from header info
  const size_t nchans = header.nchans;
  std::vector<double> frequency_table(nchans);
  float fch1 = header.fch1;
  float foff = header.foff;

  for (size_t i = 0; i < nchans; ++i) {
    frequency_table[i] = fch1 + i * foff;
  }

  float freq_min = frequency_table[0];
  float freq_max = frequency_table[nchans - 1];

  // Validate parameters
  if (freq_start < freq_min || freq_end > freq_max) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Frequency range [%.3f-%.3f MHz] out of spectrum range "
             "[%.3f-%.3f MHz]",
             freq_start, freq_end, freq_min, freq_max);
    throw std::invalid_argument(error_msg);
  }

  size_t chan_start = static_cast<size_t>((freq_start - freq_min) /
                                          (freq_max - freq_min) * (nchans - 1));
  size_t chan_end = static_cast<size_t>((freq_end - freq_min) /
                                        (freq_max - freq_min) * (nchans - 1));

  chan_start = std::max(static_cast<size_t>(0), chan_start);
  chan_end = std::min(static_cast<size_t>(nchans) - 1, chan_end);

  if (chan_start >= nchans || chan_end >= nchans) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid channel range [%zu-%zu] for %zu channels", chan_start,
             chan_end, nchans);
    throw std::invalid_argument(error_msg);
  }

  const size_t dm_steps = static_cast<size_t>((dm_high - dm_low) / dm_step) + 1;
  const float ref_freq_value =
      ref_freq ? frequency_table[chan_end] : frequency_table[chan_start];

  // Calculate the full downsampled time dimensions
  const size_t down_ndata = (header.ndata + time_downsample - 1) / time_downsample;

  // Allocate and initialize delay table on GPU
  int *d_delay_table;
  CHECK_CUDA(hipMallocManaged(
      &d_delay_table, dm_steps * (chan_end - chan_start + 1) * sizeof(int)));

  double *d_freq_table;
  CHECK_CUDA(hipMallocManaged(&d_freq_table, nchans * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_freq_table, frequency_table.data(),
                        nchans * sizeof(double), hipMemcpyHostToDevice));

  // Calculate dedispersion delay table
  dim3 block_size(64, 16);
  dim3 grid_size((dm_steps + block_size.x - 1) / block_size.x,
                 (chan_end - chan_start + 1 + block_size.y - 1) / block_size.y);

  pre_calculate_dedispersion_kernel<<<grid_size, block_size>>>(
      d_delay_table, dm_low, dm_high, dm_step, chan_start, chan_end,
      d_freq_table, ref_freq_value, header.tsamp);

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(data, header.nchans, header.ndata);

  T *d_input;
  T *d_binned_input; // 存储分bin后的数据
  CHECK_CUDA(hipMalloc(&d_input, header.ndata * nchans * sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_input, data, header.ndata * nchans * sizeof(T),
                        hipMemcpyHostToDevice));

  if (time_downsample > 1) {
    // 需要进行时间分bin降采样
    CHECK_CUDA(hipMalloc(&d_binned_input, down_ndata * nchans * sizeof(T)));
    
    printf("Performing time binning: %zu -> %zu time samples (factor %d)\n", 
           header.ndata, down_ndata, time_downsample);
    
    // 使用1D grid配置来避免grid大小限制问题
    const size_t total_elements = nchans * down_ndata;
    const int threads_per_block = 256;
    const size_t blocks_needed = (total_elements + threads_per_block - 1) / threads_per_block;
    
    printf("Binning kernel config: %zu total elements, %zu blocks, %d threads per block\n", 
           total_elements, blocks_needed, threads_per_block);
    
    auto binning_start = std::chrono::high_resolution_clock::now();
    time_binning_kernel<T><<<blocks_needed, threads_per_block>>>(
        d_binned_input, d_input, nchans, header.ndata, time_downsample, down_ndata);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    auto binning_end = std::chrono::high_resolution_clock::now();
    auto binning_duration = std::chrono::duration_cast<std::chrono::milliseconds>(binning_end - binning_start);
    printf("Time binning completed in %lld ms\n", binning_duration.count());
    
    // 释放原始输入数据
    CHECK_CUDA(hipFree(d_input));
  } else {
    // 不需要分bin，直接使用原始数据
    printf("No time binning needed (factor = 1)\n");
    d_binned_input = d_input;
  }

  printf("Processing full data: DM steps = %zu, Time samples = %zu\n", dm_steps, down_ndata);

  uint64_t *d_output;
  CHECK_CUDA(hipMalloc(&d_output, dm_steps * down_ndata * sizeof(uint64_t)));
  CHECK_CUDA(hipMemset(d_output, 0, dm_steps * down_ndata * sizeof(uint64_t)));

  int THREADS_PER_BLOCK = 256;
  dim3 threads(THREADS_PER_BLOCK);
  dim3 grids((down_ndata + threads.x - 1) / threads.x, dm_steps);

  if (use_shared_memory) {
    // Calculate shared memory size needed
    size_t max_shared_mem = device_prop.sharedMemPerBlock;
    size_t shared_mem_size = std::min(max_shared_mem / sizeof(T), 
                                     (chan_end - chan_start + 1) * THREADS_PER_BLOCK);
    
    // Ensure we don't exceed shared memory limits
    size_t actual_shared_mem = shared_mem_size * sizeof(T);
    
    printf("Using shared memory kernel with %zu bytes of shared memory\n", actual_shared_mem);
    
    dedispersion_shared_memory_kernel<T><<<grids, threads, actual_shared_mem>>>(
        d_output, d_binned_input, d_delay_table, dm_steps, time_downsample, down_ndata,
        nchans, chan_start, chan_end, 0, shared_mem_size);
  } else {
    printf("Using global memory kernel\n");
    dedispersion_kernel<T><<<grids, threads>>>(
        d_output, d_binned_input, d_delay_table, dm_steps, down_ndata, time_downsample,
        nchans, chan_start, chan_end, 0);
  }

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  // Copy back the full result
  auto dm_array = std::shared_ptr<uint64_t[]>(
      new (std::align_val_t{4096}) uint64_t[dm_steps * down_ndata](),
      [](uint64_t *p) { operator delete[](p, std::align_val_t{4096}); });

  CHECK_CUDA(hipMemcpy(dm_array.get(), d_output,
                        dm_steps * down_ndata * sizeof(uint64_t),
                        hipMemcpyDeviceToHost));

  // Clean up GPU resources
  if (time_downsample > 1) {
    CHECK_CUDA(hipFree(d_binned_input));
  } else {
    CHECK_CUDA(hipFree(d_input)); // d_binned_input == d_input when no binning
  }
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay_table));
  CHECK_CUDA(hipFree(d_freq_table));

  // Create single large dedisperseddata with all time samples
  dedisperseddata result;
  result.dm_times.emplace_back(std::move(dm_array));
  result.dm_low = dm_low;
  result.dm_high = dm_high;
  result.dm_step = dm_step;
  result.tsample = (time_downsample > 1) ? header.tsamp * time_downsample : header.tsamp; // 只有分bin时才更新时间分辨率
  result.filname = header.filename;
  result.dm_ndata = dm_steps;
  result.downtsample_ndata = down_ndata;
  result.shape = {dm_steps, down_ndata};

  printf("Full dedispersion completed. Now applying preprocessing with slicing...\n");
  Header updated_header = header;
  updated_header.tsamp = (time_downsample > 1) ? header.tsamp * time_downsample : header.tsamp; // 只有分bin时才更新时间分辨率
  return preprocess_dedisperseddata_with_slicing(result, updated_header, 1, t_sample); // time_downsample设为1，因为已经分bin完成或不需要分bin

}

template dedisperseddata_uint8
dedispered_fil_cuda<uint8_t>(Filterbank &fil, float dm_low, float dm_high,
                             float freq_start, float freq_end, float dm_step,
                             int ref_freq, int time_downsample, float t_sample, 
                             std::string mask_file, bool use_shared_memory);

template dedisperseddata_uint8
dedispered_fil_cuda<uint16_t>(Filterbank &fil, float dm_low, float dm_high,
                              float freq_start, float freq_end, float dm_step,
                              int ref_freq, int time_downsample,
                              float t_sample, std::string mask_file, bool use_shared_memory);

template dedisperseddata_uint8
dedispered_fil_cuda<uint32_t>(Filterbank &fil, float dm_low, float dm_high,
                              float freq_start, float freq_end, float dm_step,
                              int ref_freq, int time_downsample,
                              float t_sample, std::string mask_file, bool use_shared_memory);

template dedisperseddata_uint8
dedisperse_spec<uint8_t>(uint8_t *data, Header header, float dm_low,
                         float dm_high, float freq_start, float freq_end,
                         float dm_step, int ref_freq, int time_downsample,
                         float t_sample, std::string mask_file, bool use_shared_memory);

template dedisperseddata_uint8
dedisperse_spec<uint16_t>(uint16_t *data, Header header, float dm_low,
                          float dm_high, float freq_start, float freq_end,
                          float dm_step, int ref_freq, int time_downsample,
                          float t_sample, std::string mask_file, bool use_shared_memory);
                          
template dedisperseddata_uint8
dedisperse_spec<uint32_t>(uint32_t *data, Header header, float dm_low,
                          float dm_high, float freq_start, float freq_end,
                          float dm_step, int ref_freq, int time_downsample,
                          float t_sample, std::string mask_file, bool use_shared_memory);

} // namespace gpucal
