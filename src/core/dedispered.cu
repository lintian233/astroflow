#include "hip/hip_runtime.h"
#include "data.h"
#include "gpucal.h"
#include "marcoutils.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <hip/hip_vector_types.h>
#include "rfimarker.h"
#include <chrono>
#include <cstring>
#include <cstdio>
#include <cmath>
#include <limits>
#include <memory>
#include <algorithm>
#include <iostream>
#include <vector>

#ifndef AF_USE_SUBBAND
#define AF_USE_SUBBAND 1
#endif

#ifndef AF_SUBBAND_SIZE_CH
#define AF_SUBBAND_SIZE_CH 32
#endif

#ifndef AF_SUBBAND_NDM0
#define AF_SUBBAND_NDM0 32
#endif

#ifndef AF_SUBBAND_TBLOCK
#define AF_SUBBAND_TBLOCK 81920
#endif

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << " at "         \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  } while(0)

namespace gpucal {

template<typename T>
__device__ __forceinline__ uint4 load_vector4(const T* ptr) {
    if constexpr (sizeof(T) == 1)      return *reinterpret_cast<const uint4*>(ptr);
    else if constexpr (sizeof(T) == 2) { uint2 v = *reinterpret_cast<const uint2*>(ptr);
      return make_uint4(v.x, v.y, 0, 0); }
    else { uint v = *reinterpret_cast<const uint*>(ptr);
      return make_uint4(v, 0, 0, 0); }
}

template<typename T>
__device__ __forceinline__ uint64_t extract_and_sum_vector4(uint4 vec) {
    if constexpr (sizeof(T) == 1) {
        uint64_t s = 0;
        s += (vec.x & 0xFF) + ((vec.x>>8)&0xFF) + ((vec.x>>16)&0xFF) + ((vec.x>>24)&0xFF);
        s += (vec.y & 0xFF) + ((vec.y>>8)&0xFF) + ((vec.y>>16)&0xFF) + ((vec.y>>24)&0xFF);
        s += (vec.z & 0xFF) + ((vec.z>>8)&0xFF) + ((vec.z>>16)&0xFF) + ((vec.z>>24)&0xFF);
        s += (vec.w & 0xFF) + ((vec.w>>8)&0xFF) + ((vec.w>>16)&0xFF) + ((vec.w>>24)&0xFF);
        return s;
    } else if constexpr (sizeof(T) == 2) {
        uint64_t s = 0;
        s += (vec.x & 0xFFFF) + ((vec.x>>16)&0xFFFF);
        s += (vec.y & 0xFFFF) + ((vec.y>>16)&0xFFFF);
        return s;
    } else {
        return vec.x;
    }
}

template<typename T>
__host__ bool should_use_optimized_kernel(const hipDeviceProp_t& p,
                                         size_t nchans, size_t dm_steps,
                                         size_t down_ndata) {
    return (p.major >= 7) && (nchans*dm_steps*down_ndata > 1000000ULL);
}


template <typename T>
__global__ void
dedispersion_shared_memory_kernel_optimized(dedispersion_output_t<T> *output, T *input, int *delay_table,
                                           size_t dm_steps, int time_downsample,
                                           size_t down_ndata, size_t nchans,
                                           size_t chan_start, size_t chan_end,
                                           size_t start, size_t shared_mem_size) {
  const size_t dmidx = blockIdx.y;
  const size_t tidx  = blockIdx.x * blockDim.x + threadIdx.x;
  if (dmidx >= dm_steps || tidx >= down_ndata) return;

  extern __shared__ char sbuf[];
  T* __restrict__ Bloc = reinterpret_cast<T*>(sbuf);

  dedispersion_output_t<T> Sl1=0, Sl2=0, Sl3=0, Sl4=0;
  const size_t Tini = start + blockIdx.x*blockDim.x;

  const size_t Dch  = (chan_end - chan_start + 1);
  const size_t Nch  = min(shared_mem_size / blockDim.x, Dch);
  const int* __restrict__ dptr = &delay_table[dmidx * Dch];

  for (size_t c = 0; c < Dch; c += Nch) {
    const size_t niter = min(Nch, Dch - c);
    const size_t tid = threadIdx.x;

    if (tid < blockDim.x) {
      #pragma unroll 4
      for (size_t off = 0; off < niter; off += 4) {
        if (off + 3 < niter) {
          const size_t ch1 = chan_start + c + off;
          const size_t ch2 = ch1 + 1;
          const size_t ch3 = ch1 + 2;
          const size_t ch4 = ch1 + 3;
          if (ch4 < chan_end) {
            const int d1 = dptr[ch1 - chan_start] / time_downsample;
            const int d2 = dptr[ch2 - chan_start] / time_downsample;
            const int d3 = dptr[ch3 - chan_start] / time_downsample;
            const int d4 = dptr[ch4 - chan_start] / time_downsample;
            const size_t t1 = Tini + tid + d1;
            const size_t t2 = Tini + tid + d2;
            const size_t t3 = Tini + tid + d3;
            const size_t t4 = Tini + tid + d4;
            T v1 = (t1 < down_ndata) ? input[ch1 + t1*nchans] : 0;
            T v2 = (t2 < down_ndata) ? input[ch2 + t2*nchans] : 0;
            T v3 = (t3 < down_ndata) ? input[ch3 + t3*nchans] : 0;
            T v4 = (t4 < down_ndata) ? input[ch4 + t4*nchans] : 0;
            Bloc[(off+0)*blockDim.x + tid] = v1;
            Bloc[(off+1)*blockDim.x + tid] = v2;
            Bloc[(off+2)*blockDim.x + tid] = v3;
            Bloc[(off+3)*blockDim.x + tid] = v4;
          }
        } else {
          for (size_t i = off; i < niter; ++i) {
            const size_t ch = chan_start + c + i;
            if (ch < chan_end) {
              const int d = dptr[ch - chan_start] / time_downsample;
              const size_t t = Tini + tid + d;
              Bloc[i*blockDim.x + tid] = (t < down_ndata) ? input[ch + t*nchans] : 0;
            }
          }
        }
      }
    }
    __syncthreads();

    if (threadIdx.x < blockDim.x) {
      #pragma unroll 8
      for (size_t l = 0; l < niter; l += 8) {
        if (l + 7 < niter) {
          const T* sp = &Bloc[l*blockDim.x + threadIdx.x];
          Sl1 += sp[0*blockDim.x]; Sl2 += sp[1*blockDim.x];
          Sl3 += sp[2*blockDim.x]; Sl4 += sp[3*blockDim.x];
          Sl1 += sp[4*blockDim.x]; Sl2 += sp[5*blockDim.x];
          Sl3 += sp[6*blockDim.x]; Sl4 += sp[7*blockDim.x];
        } else {
          for (size_t i = l; i < niter; ++i) Sl1 += Bloc[i*blockDim.x + threadIdx.x];
        }
      }
    }
    __syncthreads();
  }
  const dedispersion_output_t<T> sum = Sl1+Sl2+Sl3+Sl4;
  output[dmidx*down_ndata + tidx] = sum;
}

template <typename T>
__global__ void
dedispersion_shared_memory_kernel(dedispersion_output_t<T> *output, T *input, int *delay_table,
                                  size_t dm_steps, int time_downsample,
                                  size_t down_ndata, size_t nchans,
                                  size_t chan_start, size_t chan_end,
                                  size_t start, size_t shared_mem_size) {
  size_t dmidx = blockIdx.y;
  size_t tidx = blockIdx.x*blockDim.x + threadIdx.x;
  if (dmidx >= dm_steps || tidx >= down_ndata) return;

  extern __shared__ char sbuf[];
  T* Bloc = reinterpret_cast<T*>(sbuf);
  dedispersion_output_t<T> Sloc = 0;
  size_t Tini = start + blockIdx.x*blockDim.x;

  size_t Dch = (chan_end - chan_start + 1);
  size_t Nch = min(shared_mem_size / blockDim.x, Dch);

  for (size_t c=0; c<Dch; c+=Nch) {
    size_t niter = min(Nch, Dch - c);
    for (size_t off=0; off<niter; ++off) {
      size_t chan = chan_start + c + off;
      if (chan < chan_end && threadIdx.x < blockDim.x) {
        int odelay = delay_table[dmidx * Dch + chan - chan_start];
        size_t d = odelay / time_downsample;
        size_t t = Tini + threadIdx.x + d;
        Bloc[off*blockDim.x + threadIdx.x] = (t < down_ndata) ? input[chan + t*nchans] : 0;
      }
    }
    __syncthreads();

    for (size_t l = 0; l < niter; ++l)
      if (threadIdx.x < blockDim.x) Sloc += Bloc[l*blockDim.x + threadIdx.x];
    __syncthreads();
  }
  output[dmidx*down_ndata + tidx] = Sloc;
}

template <typename T>
__global__ void
dedispersion_kernel_optimized(dedispersion_output_t<T> *output, T *input, int *delay_table,
                             size_t dm_steps, size_t down_ndata, int time_downsample,
                             size_t nchans, size_t chan_start, size_t chan_end,
                             size_t start) {
  const size_t dmidx = blockIdx.y;
  const size_t down_idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (dmidx >= dm_steps || down_idx >= down_ndata) return;

  const size_t base_idx = down_idx + start;
  const size_t Dch = chan_end - chan_start;
  const int* __restrict__ dptr = &delay_table[dmidx * Dch];

  dedispersion_output_t<T> s1=0,s2=0,s3=0,s4=0;
  size_t chan = chan_start;
  #pragma unroll 4
  for (; chan + 7 < chan_end; chan += 8) {
    const int d0 = dptr[chan - chan_start] / time_downsample;
    const int d1 = dptr[chan + 1 - chan_start] / time_downsample;
    const int d2 = dptr[chan + 2 - chan_start] / time_downsample;
    const int d3 = dptr[chan + 3 - chan_start] / time_downsample;
    const int d4 = dptr[chan + 4 - chan_start] / time_downsample;
    const int d5 = dptr[chan + 5 - chan_start] / time_downsample;
    const int d6 = dptr[chan + 6 - chan_start] / time_downsample;
    const int d7 = dptr[chan + 7 - chan_start] / time_downsample;

    const size_t t0 = base_idx + d0, t1 = base_idx + d1;
    const size_t t2 = base_idx + d2, t3 = base_idx + d3;
    const size_t t4 = base_idx + d4, t5 = base_idx + d5;
    const size_t t6 = base_idx + d6, t7 = base_idx + d7;

    if (t0 < down_ndata) s1 += input[chan + 0 + t0*nchans];
    if (t1 < down_ndata) s2 += input[chan + 1 + t1*nchans];
    if (t2 < down_ndata) s3 += input[chan + 2 + t2*nchans];
    if (t3 < down_ndata) s4 += input[chan + 3 + t3*nchans];
    if (t4 < down_ndata) s1 += input[chan + 4 + t4*nchans];
    if (t5 < down_ndata) s2 += input[chan + 5 + t5*nchans];
    if (t6 < down_ndata) s3 += input[chan + 6 + t6*nchans];
    if (t7 < down_ndata) s4 += input[chan + 7 + t7*nchans];
  }
  for (; chan < chan_end; ++chan) {
    const int d = dptr[chan - chan_start] / time_downsample;
    const size_t tt = base_idx + d;
    if (tt < down_ndata) s1 += input[chan + tt*nchans];
  }
  output[dmidx*down_ndata + down_idx] = s1+s2+s3+s4;
}

template <typename T>
__global__ void
dedispersion_kernel(dedispersion_output_t<T> *output, T *input, int *delay_table,
                    size_t dm_steps, size_t down_ndata, int time_downsample,
                    size_t nchans, size_t chan_start, size_t chan_end,
                    size_t start) {
  size_t dmidx = blockIdx.y;
  size_t down_idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (dmidx >= dm_steps || down_idx >= down_ndata) return;

  size_t base_idx = down_idx + start;
  dedispersion_output_t<T> sum = 0;
  for (size_t chan = chan_start; chan < chan_end; ++chan) {
    int odelay = delay_table[dmidx * (chan_end - chan_start + 1) + chan - chan_start];
    size_t d = odelay / time_downsample;
    size_t tt = base_idx + d;
    if (tt < down_ndata) sum += input[chan + tt*nchans];
  }
  output[dmidx*down_ndata + down_idx] = sum;
}

__global__ void
pre_calculate_dedispersion_kernel(int *delay_table, float dm_low, float dm_high,
                                  float dm_step, size_t chan_start,
                                  size_t chan_end, double *freq_table,
                                  float ref_freq_value, double tsamp) {
  size_t dmidx = blockDim.x*blockIdx.x + threadIdx.x;
  float dm = dm_low + (blockDim.x*blockIdx.x + threadIdx.x) * dm_step;
  if (dm > dm_high) return;
  size_t chan = blockDim.y*blockIdx.y + threadIdx.y + chan_start;
  if (chan > chan_end) return;

  double f = freq_table[chan];
  float ref2  = ref_freq_value * ref_freq_value;
  float f2    = static_cast<float>(f*f);
  float delay = 4148.808f * dm * (1.0f/f2 - 1.0f/ref2);
  delay_table[dmidx * (chan_end - chan_start + 1) + chan - chan_start] =
      static_cast<int>(roundf(delay / tsamp));
}

template <typename T>
__global__ void
time_binning_kernel(T *output, T *input, size_t nchans, size_t ndata,
                   int time_downsample, size_t down_ndata) {
  size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
  size_t total = nchans * down_ndata;
  if (idx >= total) return;

  size_t chan = idx % nchans;
  size_t down_idx = idx / nchans;

  size_t t0 = down_idx * static_cast<size_t>(time_downsample);
  size_t t1 = min(t0 + static_cast<size_t>(time_downsample), ndata);

  uint64_t sum = 0;
  constexpr uint64_t vmax = static_cast<uint64_t>(std::numeric_limits<T>::max());
  size_t cnt = t1 - t0;
  for (size_t t = t0; t < t1; ++t)
    sum += static_cast<uint64_t>(input[chan + t*nchans]);

  T result;
  // #define USE_AVERAGE_BINNING
  #define USE_SQRT_COMPRESSION
  #ifdef USE_AVERAGE_BINNING
    result = static_cast<T>(sum / (cnt ? cnt : 1));
  #elif defined(USE_RANDOM_SAMPLING)
    result = input[chan + t0*nchans];
  #elif defined(USE_SQRT_COMPRESSION)
  {
    double s = std::sqrt(static_cast<double>(sum));
    double smax = std::sqrt(static_cast<double>(vmax * (cnt ? cnt : 1)));
    double v = (s / (smax > 0.0 ? smax : 1.0)) * vmax;
    result = static_cast<T>(min(v, static_cast<double>(vmax)));
  }
  #else
    result = static_cast<T>(min(sum, vmax));
  #endif

  output[chan + down_idx*nchans] = result;
}


__global__ void divide_delay_inplace_kernel(int* delay, size_t n, int div) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) delay[i] /= div;
}

// 在 GPU 上并行计算 residual2[dm_steps, NSB]
__global__ void compute_residual2_kernel(
    int* __restrict__ residual2,
    const double* __restrict__ sbfreq,   // [NSB]
    size_t dm_steps, size_t NSB, size_t NDM0,
    float dm_low, float dm_step,
    float ref_freq_value,
    double tsamp, int time_downsample)
{
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t total = dm_steps * NSB;
  if (idx >= total) return;

  size_t m  = idx / NSB;
  size_t sb = idx % NSB;

  size_t j = m / NDM0;
  float dmj = dm_low + static_cast<float>(j * NDM0) * dm_step; // nominal DM
  float dmm = dm_low + static_cast<float>(m) * dm_step;        // final DM
  float dmdelta = dmm - dmj;

  double f = sbfreq[sb];
  float ref2 = ref_freq_value * ref_freq_value;
  float f2   = static_cast<float>(f * f);
  float delay = 4148.808f * dmdelta * (1.0f/f2 - 1.0f/ref2);

  float bins_f = delay / static_cast<float>(tsamp * time_downsample);
  int   bins   = static_cast<int>(floorf(bins_f + 0.5f)); 
  residual2[idx] = bins;
}


template <typename Tin, typename AccT>
__global__ void subband_stage1_kernel(
    AccT* __restrict__ inter,
    const Tin* __restrict__ input,
    const int* __restrict__ delay1,       
    size_t NDM_nom, size_t NSB, size_t subband_size,
    size_t nchans, size_t chan_start, size_t chan_end,
    size_t time_downsample,                // 已不再使用，仅为接口兼容
    size_t down_ndata,
    size_t t_offset,
    size_t tile1_len,
    size_t Dch)
{
  (void)time_downsample;

  const size_t t  = blockIdx.x*blockDim.x + threadIdx.x;
  const size_t j  = blockIdx.y;      
  const size_t sb = blockIdx.z;    
  if (t >= tile1_len || j >= NDM_nom || sb >= NSB) return;

  size_t ch0 = chan_start + sb*subband_size;
  size_t ch1 = min(ch0 + subband_size, chan_end + 1);

  AccT sum = 0;
  const int* dptr = &delay1[j * Dch];
  for (size_t ch = ch0; ch < ch1; ++ch) {
    int d = dptr[ch - chan_start];      
    size_t tt = t_offset + t + static_cast<size_t>(d);
    if (tt < down_ndata) sum += input[ch + tt*nchans];
  }
  inter[(j*NSB + sb)*tile1_len + t] = sum;
}

template <typename AccT>
__global__ void subband_stage2_kernel(
    AccT* __restrict__ output,
    const AccT* __restrict__ inter,
    const int*  __restrict__ residual2,     
    size_t NDM, size_t NDM0, size_t NSB,
    size_t down_ndata,
    size_t t_offset,
    size_t tile_len,
    size_t tile1_len)
{
  const size_t t   = blockIdx.x*blockDim.x + threadIdx.x;
  const size_t dm  = blockIdx.y;
  if (t >= tile_len || dm >= NDM) return;

  const size_t j = dm / NDM0;    
  AccT sum = 0;

  const int* rptr = &residual2[dm * NSB];
  const AccT* inter_base = &inter[(j * NSB) * tile1_len];

  for (size_t sb=0; sb<NSB; ++sb) {
    int dt = rptr[sb];
    size_t ti = t + static_cast<size_t>(dt);
    if (ti < tile1_len) sum += inter_base[sb*tile1_len + ti];
  }
  output[dm*down_ndata + (t_offset + t)] = sum;
}



template <typename T>
dedisperseddata_uint8 dedispered_fil_cuda(Filterbank &fil, float dm_low,
                                    float dm_high, float freq_start,
                                    float freq_end, float dm_step, int ref_freq,
                                    int time_downsample, float t_sample, int target_id,
                                    std::string mask_file) {
  // Timing variables
  hipEvent_t start_event, stop_event, stage1_start, stage1_stop, stage2_start, stage2_stop;
  float ms = 0.0f;
  float stage1_ms = 0.0f, stage2_ms = 0.0f, total_ms = 0.0f;
  std::chrono::high_resolution_clock::time_point cpu_start, cpu_end, total_start, total_end;
  float cpu_ms = 0.0f;

  int device_count; CHECK_CUDA(hipGetDeviceCount(&device_count));
  if (device_count == 0) throw std::runtime_error("No CUDA devices found");

  int device_id = target_id;
  hipDeviceProp_t device_prop;
  CHECK_CUDA(hipGetDeviceProperties(&device_prop, device_id));
  printf("Using device %d: %s\n", device_id, device_prop.name);
  CHECK_CUDA(hipSetDevice(device_id));

  // Create CUDA events for timing
  CHECK_CUDA(hipEventCreate(&start_event));
  CHECK_CUDA(hipEventCreate(&stop_event));
  CHECK_CUDA(hipEventCreate(&stage1_start));
  CHECK_CUDA(hipEventCreate(&stage1_stop));
  CHECK_CUDA(hipEventCreate(&stage2_start));
  CHECK_CUDA(hipEventCreate(&stage2_stop));

  float fil_freq_min = fil.frequency_table[0];
  float fil_freq_max = fil.frequency_table[fil.nchans - 1];

  if (freq_start < fil_freq_min || freq_end > fil_freq_max) {
    char msg[256];
    snprintf(msg, sizeof(msg),
             "Frequency range [%.3f-%.3f MHz] out of filterbank range [%.3f-%.3f MHz]",
             freq_start, freq_end, fil_freq_min, fil_freq_max);
    throw std::invalid_argument(msg);
  }

  size_t chan_start = static_cast<size_t>((freq_start - fil_freq_min) /
                          (fil_freq_max - fil_freq_min) * (fil.nchans - 1));
  size_t chan_end   = static_cast<size_t>((freq_end   - fil_freq_min) /
                          (fil_freq_max - fil_freq_min) * (fil.nchans - 1));
  chan_start = std::max<size_t>(0, chan_start);
  chan_end   = std::min<size_t>(fil.nchans - 1, chan_end);
  if (chan_start >= fil.nchans || chan_end >= fil.nchans) {
    char msg[256]; snprintf(msg, sizeof(msg),
             "Invalid channel range [%zu-%zu] for %d channels",
             chan_start, chan_end, fil.nchans);
    throw std::invalid_argument(msg);
  }

  const size_t nchans   = fil.nchans;
  const size_t dm_steps = static_cast<size_t>((dm_high - dm_low) / dm_step) + 1;
  const float ref_freq_value = ref_freq ? fil.frequency_table[chan_end]
                                        : fil.frequency_table[chan_start];
  const size_t down_ndata = (fil.ndata + time_downsample - 1) / time_downsample;

  double *d_freq_table;
  CHECK_CUDA(hipMallocManaged(&d_freq_table, nchans * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_freq_table, fil.frequency_table,
                        nchans*sizeof(double), hipMemcpyHostToDevice));

#if AF_USE_SUBBAND
  T *h_input = static_cast<T*>(fil.data);
  T *d_input = nullptr;
  // Total timing start
  total_start = std::chrono::high_resolution_clock::now();
  // Host to Device copy timing
  CHECK_CUDA(hipEventRecord(start_event));
  CHECK_CUDA(hipMalloc(&d_input, fil.ndata * nchans * sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_input, h_input,
                        fil.ndata*nchans*sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA(hipEventRecord(stop_event));
  CHECK_CUDA(hipEventSynchronize(stop_event));
  CHECK_CUDA(hipEventElapsedTime(&ms, start_event, stop_event));
  printf("[TIMER] Host to Device copy: %.3f ms\n", ms);

  // 时间降采样
  T *d_binned_input = d_input;
  if (time_downsample > 1) {
    CHECK_CUDA(hipMalloc(&d_binned_input, down_ndata * nchans * sizeof(T)));
    const size_t total = nchans * down_ndata;
    const int TPB = 256;
    const size_t nblk = (total + TPB - 1)/TPB;
    CHECK_CUDA(hipEventRecord(start_event));
    time_binning_kernel<T><<<nblk, TPB>>>(d_binned_input, d_input,
        nchans, fil.ndata, time_downsample, down_ndata);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventRecord(stop_event));
    CHECK_CUDA(hipEventSynchronize(stop_event));
    CHECK_CUDA(hipEventElapsedTime(&ms, start_event, stop_event));
    printf("[TIMER] Time binning kernel: %.3f ms\n", ms);
    CHECK_CUDA(hipFree(d_input));
  }

  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(d_binned_input, nchans, down_ndata);

  const size_t Dch = (chan_end - chan_start + 1);
  const size_t NSB = (Dch + AF_SUBBAND_SIZE_CH - 1) / AF_SUBBAND_SIZE_CH;
  const size_t NDM0 = AF_SUBBAND_NDM0;
  const size_t NDM_nom = (dm_steps + NDM0 - 1) / NDM0;

  int *d_delay1 = nullptr;
  CHECK_CUDA(hipMallocManaged(&d_delay1, NDM_nom * Dch * sizeof(int)));
  {
    float dm_step_coarse = dm_step * static_cast<float>(NDM0);
    float dm_high_coarse = dm_low + (NDM_nom - 1) * dm_step_coarse;

    dim3 bs(64, 16);
    dim3 gs((NDM_nom + bs.x - 1)/bs.x,
            (Dch      + bs.y - 1)/bs.y);
    pre_calculate_dedispersion_kernel<<<gs, bs>>>(
      d_delay1, dm_low, dm_high_coarse, dm_step_coarse,
      chan_start, chan_end, d_freq_table, ref_freq_value, fil.tsamp);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }

  {
    size_t tot = static_cast<size_t>(NDM_nom) * Dch;
    const int TPB_DIV = 256;
    size_t nblk_div = (tot + TPB_DIV - 1) / TPB_DIV;
    divide_delay_inplace_kernel<<<nblk_div, TPB_DIV>>>(d_delay1, tot, time_downsample);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }


  std::vector<double> h_sbfreq(NSB, 0.0);
  for (size_t sb=0; sb<NSB; ++sb) {
    size_t ch0 = chan_start + sb*AF_SUBBAND_SIZE_CH;
    size_t ch1 = std::min(ch0 + AF_SUBBAND_SIZE_CH, chan_end + 1);
    size_t mid = (ch0 + ch1 - 1) / 2;
    h_sbfreq[sb] = fil.frequency_table[mid];
  }


  double* d_sbfreq = nullptr;
  CHECK_CUDA(hipMalloc(&d_sbfreq, NSB * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_sbfreq, h_sbfreq.data(),
                        NSB * sizeof(double), hipMemcpyHostToDevice));

  int *d_residual2 = nullptr;
  CHECK_CUDA(hipMalloc(&d_residual2, dm_steps * NSB * sizeof(int)));
  {
    const size_t total = dm_steps * NSB;
    const int TPB = 256;
    const size_t nblk = (total + TPB - 1) / TPB;
    compute_residual2_kernel<<<nblk, TPB>>>(
        d_residual2, d_sbfreq,
        dm_steps, NSB, NDM0,
        dm_low, dm_step, ref_freq_value,
        fil.tsamp, time_downsample);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }

  int max_residual = 0;
  {
    void* d_tmp = nullptr; size_t tmp_bytes = 0;
    int* d_max = nullptr;
    const size_t N = dm_steps * NSB;
    hipcub::DeviceReduce::Max(d_tmp, tmp_bytes, d_residual2, d_max, N);
    CHECK_CUDA(hipMalloc(&d_tmp, tmp_bytes));
    CHECK_CUDA(hipMalloc(&d_max, sizeof(int)));
    hipcub::DeviceReduce::Max(d_tmp, tmp_bytes, d_residual2, d_max, N);
    CHECK_CUDA(hipMemcpy(&max_residual, d_max, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(d_tmp));
    CHECK_CUDA(hipFree(d_max));
  }
  CHECK_CUDA(hipFree(d_sbfreq));

  dedispersion_output_t<T> *d_output = nullptr;
  CHECK_CUDA(hipMalloc(&d_output, dm_steps * down_ndata * sizeof(dedispersion_output_t<T>)));
  CHECK_CUDA(hipMemset(d_output, 0, dm_steps * down_ndata * sizeof(dedispersion_output_t<T>)));

  const size_t TPB = 256;

  for (size_t t0 = 0; t0 < down_ndata; t0 += AF_SUBBAND_TBLOCK) {
    const size_t tile_len  = std::min(static_cast<size_t>(AF_SUBBAND_TBLOCK), down_ndata - t0);
    const size_t tile1_len = std::min(tile_len + static_cast<size_t>(max_residual), down_ndata - t0);

    dedispersion_output_t<T> *d_inter = nullptr;
    CHECK_CUDA(hipMalloc(&d_inter, NDM_nom*NSB*tile1_len*sizeof(dedispersion_output_t<T>)));

    // Dedispersion stage 1 timing

    CHECK_CUDA(hipEventRecord(stage1_start));
    {
      dim3 grid1((tile1_len + TPB - 1)/TPB, NDM_nom, NSB);
      subband_stage1_kernel<T, dedispersion_output_t<T>><<<grid1, TPB>>>(
        d_inter, d_binned_input, d_delay1,
        NDM_nom, NSB, AF_SUBBAND_SIZE_CH,
        nchans, chan_start, chan_end, time_downsample, down_ndata,
        t0, tile1_len, Dch);
      CHECK_CUDA(hipGetLastError());
    }
    CHECK_CUDA(hipEventRecord(stage1_stop));
    CHECK_CUDA(hipEventSynchronize(stage1_stop));
    CHECK_CUDA(hipEventElapsedTime(&ms, stage1_start, stage1_stop));
    stage1_ms += ms;

    // Dedispersion stage 2 timing
    CHECK_CUDA(hipEventRecord(stage2_start));
    {
      dim3 grid2((tile_len + TPB - 1)/TPB, dm_steps);
      subband_stage2_kernel<dedispersion_output_t<T>><<<grid2, TPB>>>(
        d_output, d_inter, d_residual2,
        dm_steps, NDM0, NSB,
        down_ndata, t0, tile_len, tile1_len);
      CHECK_CUDA(hipGetLastError());
    }
    CHECK_CUDA(hipEventRecord(stage2_stop));
    CHECK_CUDA(hipEventSynchronize(stage2_stop));
    CHECK_CUDA(hipEventElapsedTime(&ms, stage2_start, stage2_stop));
    stage2_ms += ms;

    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(d_inter));
  }

  auto t_total1 = std::chrono::high_resolution_clock::now();
  auto dm_array_typed = std::shared_ptr<dedispersion_output_t<T>[]>( new (std::align_val_t{4096})
      dedispersion_output_t<T>[dm_steps * down_ndata](),
      [](dedispersion_output_t<T>* p){ operator delete[](p, std::align_val_t{4096}); });
  auto t_total2 = std::chrono::high_resolution_clock::now();
  cpu_ms = std::chrono::duration<float, std::milli>(t_total2 - t_total1).count();
  printf("[TIMER] allocate pinned host memory time: %.3f ms\n", cpu_ms);


  // Device to Host copy timing
  CHECK_CUDA(hipEventRecord(start_event));
  CHECK_CUDA(hipMemcpy(dm_array_typed.get(), d_output,
                        dm_steps*down_ndata*sizeof(dedispersion_output_t<T>),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipEventRecord(stop_event));
  CHECK_CUDA(hipEventSynchronize(stop_event));
  CHECK_CUDA(hipEventElapsedTime(&ms, start_event, stop_event));
  printf("[TIMER] Device to Host copy: %.3f ms\n", ms);

  // Total timing end
  total_end = std::chrono::high_resolution_clock::now();
  total_ms = std::chrono::duration<float, std::milli>(total_end - total_start).count();

  printf("[TIMER] Dedispersion Stage 1 total: %.3f ms\n", stage1_ms);
  printf("[TIMER] Dedispersion Stage 2 total: %.3f ms\n", stage2_ms);
  printf("[TIMER] Total elapsed (including all stages and copies): %.3f ms\n", total_ms);

  // Destroy CUDA events
  CHECK_CUDA(hipEventDestroy(start_event));
  CHECK_CUDA(hipEventDestroy(stop_event));
  CHECK_CUDA(hipEventDestroy(stage1_start));
  CHECK_CUDA(hipEventDestroy(stage1_stop));
  CHECK_CUDA(hipEventDestroy(stage2_start));
  CHECK_CUDA(hipEventDestroy(stage2_stop));

  auto t0 = std::chrono::high_resolution_clock::now();
  if (time_downsample > 1) CHECK_CUDA(hipFree(d_binned_input));
  else                     CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay1));
  CHECK_CUDA(hipFree(d_residual2));
  CHECK_CUDA(hipFree(d_freq_table));
  auto t1 = std::chrono::high_resolution_clock::now();
  cpu_ms = std::chrono::duration<float, std::milli>(t1 - t0).count();
  printf("[TIMER] free memory time: %.3f ms\n", cpu_ms);

  DedispersedDataTyped<dedispersion_output_t<T>> typed_result;
  typed_result.dm_times.emplace_back(std::move(dm_array_typed));
  typed_result.dm_low = dm_low;
  typed_result.dm_high = dm_high;
  typed_result.dm_step = dm_step;
  typed_result.tsample = (time_downsample > 1) ? fil.tsamp * time_downsample : fil.tsamp;
  typed_result.filname = fil.filename;
  typed_result.dm_ndata = dm_steps;
  typed_result.downtsample_ndata = down_ndata;
  typed_result.shape = {dm_steps, down_ndata};

  Header temp_header;
  temp_header.tsamp = typed_result.tsample;
  temp_header.filename = fil.filename;

  return preprocess_typed_dedisperseddata_with_slicing<T>(typed_result, temp_header, 1, t_sample);
#else

  int *d_delay_table;
  CHECK_CUDA(hipMallocManaged(
      &d_delay_table, dm_steps * (chan_end - chan_start + 1) * sizeof(int)));

  dim3 block_size(64, 16);
  dim3 grid_size((dm_steps + block_size.x - 1) / block_size.x,
                 (chan_end - chan_start + 1 + block_size.y - 1) / block_size.y);

  pre_calculate_dedispersion_kernel<<<grid_size, block_size>>>(
      d_delay_table, dm_low, dm_high, dm_step, chan_start, chan_end,
      d_freq_table, ref_freq_value, fil.tsamp);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  T *d_input;
  T *d_binned_input;
  T *data = static_cast<T *>(fil.data);

  CHECK_CUDA(hipMalloc(&d_input, fil.ndata * nchans * sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_input, data, fil.ndata * nchans * sizeof(T), hipMemcpyHostToDevice));

  if (time_downsample > 1) {
    CHECK_CUDA(hipMalloc(&d_binned_input, down_ndata * nchans * sizeof(T)));
    const size_t total_elements = nchans * down_ndata;
    const int threads_per_block = 256;
    const size_t blocks_needed = (total_elements + threads_per_block - 1) / threads_per_block;
    time_binning_kernel<T><<<blocks_needed, threads_per_block>>>(
        d_binned_input, d_input, nchans, fil.ndata, time_downsample, down_ndata);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(d_input));
  } else {
    d_binned_input = d_input;
  }

  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(d_binned_input, nchans, down_ndata);

  dedispersion_output_t<T> *d_output;
  CHECK_CUDA(hipMalloc(&d_output, dm_steps * down_ndata * sizeof(dedispersion_output_t<T>)));
  CHECK_CUDA(hipMemset(d_output, 0, dm_steps * down_ndata * sizeof(dedispersion_output_t<T>)));

  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(d_binned_input, nchans, down_ndata);

  int THREADS_PER_BLOCK = 256;
  dim3 threads(THREADS_PER_BLOCK);
  dim3 grids((down_ndata + threads.x - 1) / threads.x, dm_steps);

  bool use_optimized = should_use_optimized_kernel<T>(device_prop, nchans, dm_steps, down_ndata);
  bool use_shared_memory = true;

  if (use_shared_memory) {
    size_t max_shared_mem = device_prop.sharedMemPerBlock;
    size_t shared_mem_size = std::min(max_shared_mem / sizeof(T),
                                     (chan_end - chan_start + 1) * (size_t)THREADS_PER_BLOCK);
    size_t actual_shared_mem = shared_mem_size * sizeof(T);
    if (use_optimized) {
      dedispersion_shared_memory_kernel_optimized<T><<<grids, threads, actual_shared_mem>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, time_downsample, down_ndata,
          nchans, chan_start, chan_end, 0, shared_mem_size);
    } else {
      dedispersion_shared_memory_kernel<T><<<grids, threads, actual_shared_mem>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, time_downsample, down_ndata,
          nchans, chan_start, chan_end, 0, shared_mem_size);
    }
  } else {
    if (use_optimized) {
      dedispersion_kernel_optimized<T><<<grids, threads>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, down_ndata, time_downsample,
          nchans, chan_start, chan_end, 0);
    } else {
      dedispersion_kernel<T><<<grids, threads>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, down_ndata, time_downsample,
          nchans, chan_start, chan_end, 0);
    }
  }
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  auto dm_array_typed = std::shared_ptr<dedispersion_output_t<T>[]>( new (std::align_val_t{4096})
      dedispersion_output_t<T>[dm_steps * down_ndata](),
      [](dedispersion_output_t<T>* p){ operator delete[](p, std::align_val_t{4096}); });

  CHECK_CUDA(hipMemcpy(dm_array_typed.get(), d_output,
                        dm_steps*down_ndata*sizeof(dedispersion_output_t<T>),
                        hipMemcpyDeviceToHost));

  if (time_downsample > 1) CHECK_CUDA(hipFree(d_binned_input));
  else                     CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay_table));
  CHECK_CUDA(hipFree(d_freq_table));

  DedispersedDataTyped<dedispersion_output_t<T>> typed_result;
  typed_result.dm_times.emplace_back(std::move(dm_array_typed));
  typed_result.dm_low = dm_low;
  typed_result.dm_high = dm_high;
  typed_result.dm_step = dm_step;
  typed_result.tsample = (time_downsample > 1) ? fil.tsamp * time_downsample : fil.tsamp;
  typed_result.filname = fil.filename;
  typed_result.dm_ndata = dm_steps;
  typed_result.downtsample_ndata = down_ndata;
  typed_result.shape = {dm_steps, down_ndata};

  Header temp_header;
  temp_header.tsamp = typed_result.tsample;
  temp_header.filename = fil.filename;
  return preprocess_typed_dedisperseddata_with_slicing<T>(typed_result, temp_header, 1, t_sample);
#endif
}

template <typename T>
dedisperseddata_uint8 dedisperse_spec(T *data, Header header, float dm_low,
                                float dm_high, float freq_start, float freq_end,
                                float dm_step, int ref_freq,
                                int time_downsample, float t_sample, int target_id,
                                std::string mask_file) {

  int device_count; CHECK_CUDA(hipGetDeviceCount(&device_count));
  if (!device_count) throw std::runtime_error("No CUDA devices found");

  int device_id = target_id;
  hipDeviceProp_t device_prop;
  CHECK_CUDA(hipGetDeviceProperties(&device_prop, device_id));
  printf("Using device %d: %s\n", device_id, device_prop.name);
  CHECK_CUDA(hipSetDevice(device_id));

  const size_t nchans = header.nchans;
  std::vector<double> h_freq(nchans);
  for (size_t i=0;i<nchans;++i) h_freq[i] = header.fch1 + i*header.foff;

  float freq_min = h_freq.front();
  float freq_max = h_freq.back();
  if (freq_start < freq_min || freq_end > freq_max) {
    char msg[256];
    snprintf(msg, sizeof(msg),
             "Frequency range [%.3f-%.3f MHz] out of spectrum range [%.3f-%.3f MHz]",
             freq_start, freq_end, freq_min, freq_max);
    throw std::invalid_argument(msg);
  }

  size_t chan_start = static_cast<size_t>((freq_start - freq_min) /
                                          (freq_max - freq_min) * (nchans - 1));
  size_t chan_end   = static_cast<size_t>((freq_end   - freq_min) /
                                          (freq_max - freq_min) * (nchans - 1));
  chan_start = std::max<size_t>(0, chan_start);
  chan_end   = std::min<size_t>(nchans - 1, chan_end);
  if (chan_start >= nchans || chan_end >= nchans) {
    char msg[256]; snprintf(msg, sizeof(msg),
             "Invalid channel range [%zu-%zu] for %zu channels", chan_start, chan_end, nchans);
    throw std::invalid_argument(msg);
  }

  const size_t dm_steps = static_cast<size_t>((dm_high - dm_low) / dm_step) + 1;
  const float ref_freq_value = ref_freq ? h_freq[chan_end] : h_freq[chan_start];
  const size_t down_ndata = (header.ndata + time_downsample - 1) / time_downsample;

  double *d_freq_table;
  CHECK_CUDA(hipMallocManaged(&d_freq_table, nchans*sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_freq_table, h_freq.data(),
                        nchans*sizeof(double), hipMemcpyHostToDevice));

#if AF_USE_SUBBAND
  T *d_input=nullptr;
  CHECK_CUDA(hipMalloc(&d_input, header.ndata*nchans*sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_input, data, header.ndata*nchans*sizeof(T), hipMemcpyHostToDevice));

  T *d_binned_input = d_input;
  if (time_downsample > 1) {
    CHECK_CUDA(hipMalloc(&d_binned_input, down_ndata*nchans*sizeof(T)));
    const size_t total = nchans*down_ndata;
    const int TPB = 256; const size_t nblk = (total + TPB - 1)/TPB;
    time_binning_kernel<T><<<nblk, TPB>>>(
        d_binned_input, d_input, nchans, header.ndata, time_downsample, down_ndata);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(d_input));
  }

  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(d_binned_input, nchans, down_ndata);

  const size_t Dch  = (chan_end - chan_start + 1);
  const size_t NSB  = (Dch + AF_SUBBAND_SIZE_CH - 1) / AF_SUBBAND_SIZE_CH;
  const size_t NDM0 = AF_SUBBAND_NDM0;
  const size_t NDM_nom = (dm_steps + NDM0 - 1) / NDM0;

  int *d_delay1=nullptr;
  CHECK_CUDA(hipMallocManaged(&d_delay1, NDM_nom*Dch*sizeof(int)));
  {
    float dm_step_coarse = dm_step * static_cast<float>(NDM0);
    float dm_high_coarse = dm_low + (NDM_nom - 1)*dm_step_coarse;
    dim3 bs(64,16);
    dim3 gs((NDM_nom + bs.x - 1)/bs.x, (Dch + bs.y - 1)/bs.y);
    pre_calculate_dedispersion_kernel<<<gs, bs>>>(
      d_delay1, dm_low, dm_high_coarse, dm_step_coarse,
      chan_start, chan_end, d_freq_table, ref_freq_value, header.tsamp);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }

  {
    size_t tot = static_cast<size_t>(NDM_nom) * Dch;
    const int TPB_DIV = 256;
    size_t nblk_div = (tot + TPB_DIV - 1) / TPB_DIV;
    divide_delay_inplace_kernel<<<nblk_div, TPB_DIV>>>(d_delay1, tot, time_downsample);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }

  std::vector<double> h_sbfreq(NSB, 0.0);
  for (size_t sb=0; sb<NSB; ++sb) {
    size_t ch0 = chan_start + sb*AF_SUBBAND_SIZE_CH;
    size_t ch1 = std::min(ch0 + AF_SUBBAND_SIZE_CH, chan_end + 1);
    size_t mid = (ch0 + ch1 - 1)/2;
    h_sbfreq[sb] = h_freq[mid];
  }


  double* d_sbfreq = nullptr;
  CHECK_CUDA(hipMalloc(&d_sbfreq, NSB * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_sbfreq, h_sbfreq.data(),
                        NSB * sizeof(double), hipMemcpyHostToDevice));

  int *d_residual2=nullptr;
  CHECK_CUDA(hipMalloc(&d_residual2, dm_steps * NSB * sizeof(int)));
  {
    const size_t total = dm_steps * NSB;
    const int TPB = 256;
    const size_t nblk = (total + TPB - 1) / TPB;
    compute_residual2_kernel<<<nblk, TPB>>>(
        d_residual2, d_sbfreq,
        dm_steps, NSB, NDM0,
        dm_low, dm_step, ref_freq_value,
        header.tsamp, time_downsample);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }

  int max_residual = 0;
  {
    void* d_tmp=nullptr; size_t tmp_bytes=0;
    int* d_max=nullptr;
    const size_t N = dm_steps * NSB;
    hipcub::DeviceReduce::Max(d_tmp, tmp_bytes, d_residual2, d_max, N);
    CHECK_CUDA(hipMalloc(&d_tmp, tmp_bytes));
    CHECK_CUDA(hipMalloc(&d_max, sizeof(int)));
    hipcub::DeviceReduce::Max(d_tmp, tmp_bytes, d_residual2, d_max, N);
    CHECK_CUDA(hipMemcpy(&max_residual, d_max, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(d_tmp));
    CHECK_CUDA(hipFree(d_max));
  }
  CHECK_CUDA(hipFree(d_sbfreq));

  dedispersion_output_t<T> *d_output=nullptr;
  CHECK_CUDA(hipMalloc(&d_output, dm_steps*down_ndata*sizeof(dedispersion_output_t<T>)));
  CHECK_CUDA(hipMemset(d_output, 0, dm_steps*down_ndata*sizeof(dedispersion_output_t<T>)));

  const size_t TPB = 256;
  for (size_t t0=0; t0<down_ndata; t0 += AF_SUBBAND_TBLOCK) {
    const size_t tile_len  = std::min(static_cast<size_t>(AF_SUBBAND_TBLOCK), down_ndata - t0);
    const size_t tile1_len = std::min(tile_len + static_cast<size_t>(max_residual), down_ndata - t0);

    dedispersion_output_t<T>* d_inter=nullptr;
    CHECK_CUDA(hipMalloc(&d_inter, NDM_nom*NSB*tile1_len*sizeof(dedispersion_output_t<T>)));

    dim3 grid1((tile1_len + TPB - 1)/TPB, NDM_nom, NSB);
    subband_stage1_kernel<T, dedispersion_output_t<T>><<<grid1, TPB>>>(
      d_inter, d_binned_input, d_delay1,
      NDM_nom, NSB, AF_SUBBAND_SIZE_CH,
      nchans, chan_start, chan_end, time_downsample, down_ndata,
      t0, tile1_len, Dch);
    CHECK_CUDA(hipGetLastError());

    dim3 grid2((tile_len + TPB - 1)/TPB, dm_steps);
    subband_stage2_kernel<dedispersion_output_t<T>><<<grid2, TPB>>>(
      d_output, d_inter, d_residual2,
      dm_steps, NDM0, NSB,
      down_ndata, t0, tile_len, tile1_len);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipFree(d_inter));
  }

  auto dm_array_typed = std::shared_ptr<dedispersion_output_t<T>[]>( new (std::align_val_t{4096})
      dedispersion_output_t<T>[dm_steps * down_ndata](),
      [](dedispersion_output_t<T>* p){ operator delete[](p, std::align_val_t{4096}); });

  CHECK_CUDA(hipMemcpy(dm_array_typed.get(), d_output,
                        dm_steps*down_ndata*sizeof(dedispersion_output_t<T>),
                        hipMemcpyDeviceToHost));

  if (time_downsample > 1) CHECK_CUDA(hipFree(d_binned_input));
  else                     CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay1));
  CHECK_CUDA(hipFree(d_residual2));
  CHECK_CUDA(hipFree(d_freq_table));

  DedispersedDataTyped<dedispersion_output_t<T>> typed_result;
  typed_result.dm_times.emplace_back(std::move(dm_array_typed));
  typed_result.dm_low = dm_low;
  typed_result.dm_high = dm_high;
  typed_result.dm_step = dm_step;
  typed_result.tsample = (time_downsample > 1) ? header.tsamp * time_downsample : header.tsamp;
  typed_result.filname = header.filename;
  typed_result.dm_ndata = dm_steps;
  typed_result.downtsample_ndata = down_ndata;
  typed_result.shape = {dm_steps, down_ndata};

  Header updated = header;
  updated.tsamp = typed_result.tsample;
  return preprocess_typed_dedisperseddata_with_slicing<T>(typed_result, updated, 1, t_sample);

#else

  int *d_delay_table;
  CHECK_CUDA(hipMallocManaged(
      &d_delay_table, dm_steps * (chan_end - chan_start + 1) * sizeof(int)));

  dim3 block_size(64, 16);
  dim3 grid_size((dm_steps + block_size.x - 1) / block_size.x,
                 (chan_end - chan_start + 1 + block_size.y - 1) / block_size.y);

  pre_calculate_dedispersion_kernel<<<grid_size, block_size>>>(
      d_delay_table, dm_low, dm_high, dm_step, chan_start, chan_end,
      d_freq_table, ref_freq_value, header.tsamp);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());


  T *d_input; T *d_binned_input;
  CHECK_CUDA(hipMalloc(&d_input, header.ndata*nchans*sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_input, data, header.ndata*nchans*sizeof(T), hipMemcpyHostToDevice));

  if (time_downsample > 1) {
    CHECK_CUDA(hipMalloc(&d_binned_input, down_ndata*nchans*sizeof(T)));
    const size_t total = nchans*down_ndata;
    const int TPB = 256; const size_t nblk = (total + TPB - 1)/TPB;
    time_binning_kernel<T><<<nblk, TPB>>>(
        d_binned_input, d_input, nchans, header.ndata, time_downsample, down_ndata);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(d_input));
  } else d_binned_input = d_input;

  RfiMarker<T> rfi_marker(mask_file);
  rfi_marker.mark_rfi(d_binned_input, nchans, down_ndata);

  dedispersion_output_t<T> *d_output;
  CHECK_CUDA(hipMalloc(&d_output, dm_steps*down_ndata*sizeof(dedispersion_output_t<T>)));
  CHECK_CUDA(hipMemset(d_output, 0, dm_steps*down_ndata*sizeof(dedispersion_output_t<T>)));

  int THREADS_PER_BLOCK = 256;
  dim3 threads(THREADS_PER_BLOCK);
  dim3 grids((down_ndata + threads.x - 1) / threads.x, dm_steps);

  bool use_optimized = should_use_optimized_kernel<T>(device_prop, nchans, dm_steps, down_ndata);
  bool use_shared_memory = true;

  if (use_shared_memory) {
    size_t max_shared_mem = device_prop.sharedMemPerBlock;
    size_t shared_mem_size = std::min(max_shared_mem / sizeof(T),
                                      (chan_end - chan_start + 1) * (size_t)THREADS_PER_BLOCK);
    size_t actual_shared_mem = shared_mem_size * sizeof(T);

    if (use_optimized) {
      dedispersion_shared_memory_kernel_optimized<T><<<grids, threads, actual_shared_mem>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, time_downsample, down_ndata,
          nchans, chan_start, chan_end, 0, shared_mem_size);
    } else {
      dedispersion_shared_memory_kernel<T><<<grids, threads, actual_shared_mem>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, time_downsample, down_ndata,
          nchans, chan_start, chan_end, 0, shared_mem_size);
    }
  } else {
    if (use_optimized) {
      dedispersion_kernel_optimized<T><<<grids, threads>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, down_ndata, time_downsample,
          nchans, chan_start, chan_end, 0);
    } else {
      dedispersion_kernel<T><<<grids, threads>>>(
          d_output, d_binned_input, d_delay_table, dm_steps, down_ndata, time_downsample,
          nchans, chan_start, chan_end, 0);
    }
  }
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  auto dm_array_typed = std::shared_ptr<dedispersion_output_t<T>[]>( new (std::align_val_t{4096})
      dedispersion_output_t<T>[dm_steps * down_ndata](),
      [](dedispersion_output_t<T>* p){ operator delete[](p, std::align_val_t{4096}); });

  CHECK_CUDA(hipMemcpy(dm_array_typed.get(), d_output,
                        dm_steps*down_ndata*sizeof(dedispersion_output_t<T>),
                        hipMemcpyDeviceToHost));

  if (time_downsample > 1) CHECK_CUDA(hipFree(d_binned_input));
  else                     CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay_table));
  CHECK_CUDA(hipFree(d_freq_table));

  DedispersedDataTyped<dedispersion_output_t<T>> typed_result;
  typed_result.dm_times.emplace_back(std::move(dm_array_typed));
  typed_result.dm_low = dm_low;
  typed_result.dm_high = dm_high;
  typed_result.dm_step = dm_step;
  typed_result.tsample = (time_downsample > 1) ? header.tsamp * time_downsample : header.tsamp;
  typed_result.filname = header.filename;
  typed_result.dm_ndata = dm_steps;
  typed_result.downtsample_ndata = down_ndata;
  typed_result.shape = {dm_steps, down_ndata};

  Header updated = header;
  updated.tsamp = typed_result.tsample;
  return preprocess_typed_dedisperseddata_with_slicing<T>(typed_result, updated, 1, t_sample);
#endif
}


template dedisperseddata_uint8
dedispered_fil_cuda<uint8_t>(Filterbank &fil, float dm_low, float dm_high,
                             float freq_start, float freq_end, float dm_step,
                             int ref_freq, int time_downsample, float t_sample,
                             int target_id, std::string mask_file);

template dedisperseddata_uint8
dedispered_fil_cuda<uint16_t>(Filterbank &fil, float dm_low, float dm_high,
                              float freq_start, float freq_end, float dm_step,
                              int ref_freq, int time_downsample,
                              float t_sample, int target_id, std::string mask_file);

template dedisperseddata_uint8
dedispered_fil_cuda<uint32_t>(Filterbank &fil, float dm_low, float dm_high,
                              float freq_start, float freq_end, float dm_step,
                              int ref_freq, int time_downsample,
                              float t_sample, int target_id, std::string mask_file);

template dedisperseddata_uint8
dedisperse_spec<uint8_t>(uint8_t *data, Header header, float dm_low,
                         float dm_high, float freq_start, float freq_end,
                         float dm_step, int ref_freq, int time_downsample,
                         float t_sample, int target_id, std::string mask_file);

template dedisperseddata_uint8
dedisperse_spec<uint16_t>(uint16_t *data, Header header, float dm_low,
                          float dm_high, float freq_start, float freq_end,
                          float dm_step, int ref_freq, int time_downsample,
                          float t_sample, int target_id, std::string mask_file);

template dedisperseddata_uint8
dedisperse_spec<uint32_t>(uint32_t *data, Header header, float dm_low,
                          float dm_high, float freq_start, float freq_end,
                          float dm_step, int ref_freq, int time_downsample,
                          float t_sample, int target_id, std::string mask_file);

} // namespace gpucal
