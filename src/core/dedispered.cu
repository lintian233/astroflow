#include "hip/hip_runtime.h"
#include "data.h"
#include "gpucal.h"
#include "marcoutils.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <hip/hip_vector_types.h>

// cuda atomicAdd

#define CHECK_CUDA(call)                                                       \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << " at "         \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

namespace gpucal {
template <typename T>
__global__ void
dedispersion_kernel(uint64_t *output, T *input, int *delay_table,
                    size_t dm_steps, int time_downsample, size_t ndata,
                    size_t nchans, size_t chan_start, size_t chan_end,
                    size_t start, size_t down_ndata) {
  size_t dmidx = blockIdx.y;
  size_t down_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dmidx >= dm_steps)
    return;
  if (down_idx >= down_ndata)
    return;

  size_t base_idx = down_idx * time_downsample + start;
  uint64_t sum = 0;
  for (size_t chan = chan_start; chan < chan_end; ++chan) {
    size_t target_idx =
        base_idx +
        delay_table[dmidx * (chan_end - chan_start + 1) + chan - chan_start];
    if (target_idx > 0 && target_idx < ndata) {
      sum += input[chan + target_idx * nchans];
    } else {
      sum += 0;
    }
  }
  /* if (down_idx == 4000) {
    printf("sum %d\n", sum);
  } */
  output[dmidx * down_ndata + down_idx] = sum;
}

__global__ void
pre_calculate_dedispersion_kernel(int *delay_table, float dm_low, float dm_high,
                                  float dm_step, size_t chan_start,
                                  size_t chan_end, double *freq_table,
                                  float ref_freq_value, double tsamp) {

  size_t dmidx = blockDim.x * blockIdx.x + threadIdx.x;
  float dm = dm_low + (blockDim.x * blockIdx.x + threadIdx.x) * (dm_step);
  if (dm > dm_high)
    return;
  size_t chan = blockDim.y * blockIdx.y + threadIdx.y + chan_start;
  if (chan > chan_end)
    return;

  double freq = freq_table[chan];
  float ref_2 = ref_freq_value * ref_freq_value;
  float freq_2 = freq * freq;

  float delay = 4148.808f * dm * (1.0f / freq_2 - 1.0f / ref_2);
  delay_table[dmidx * (chan_end - chan_start + 1) + chan - chan_start] =
      static_cast<int>(roundf(delay / tsamp));
}

template <typename T>
dedisperseddata dedispered_fil_cuda(Filterbank &fil, float dm_low,
                                    float dm_high, float freq_start,
                                    float freq_end, float dm_step, int ref_freq,
                                    int time_downsample, float t_sample) {

  // get all cuda devices
  int device_count;
  CHECK_CUDA(hipGetDeviceCount(&device_count));
  if (device_count == 0) {
    throw std::runtime_error("No CUDA devices found");
  }

  int device_id;
  if (device_count == 1) {
    device_id = 0;
  } else if (device_count == 4) {
    device_id = 2;
  } else {
    device_id = 0;
  }
  // print device info
  hipDeviceProp_t device_prop;
  CHECK_CUDA(hipGetDeviceProperties(&device_prop, device_id));
  printf("Using device %d: %s\n", device_id, device_prop.name);

  CHECK_CUDA(hipSetDevice(device_id));
  // check if the device is compatible

  float fil_freq_min = fil.frequency_table[0];
  float fil_freq_max = fil.frequency_table[fil.nchans - 1];

  if (freq_start < fil_freq_min || freq_end > fil_freq_max) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Frequency range [%.3f-%.3f MHz] out of filterbank range "
             "[%.3f-%.3f MHz]",
             freq_start, freq_end, fil_freq_min, fil_freq_max);
    throw std::invalid_argument(error_msg);
  }
  size_t chan_start =
      static_cast<size_t>((freq_start - fil_freq_min) /
                          (fil_freq_max - fil_freq_min) * (fil.nchans - 1));
  size_t chan_end =
      static_cast<size_t>((freq_end - fil_freq_min) /
                          (fil_freq_max - fil_freq_min) * (fil.nchans - 1));

  chan_start = std::max(static_cast<size_t>(0), chan_start);
  chan_end = std::min(static_cast<size_t>(fil.nchans - 1), chan_end);

  if (chan_start >= fil.nchans || chan_end >= fil.nchans) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid channel range [%zu-%zu] for %d channels", chan_start,
             chan_end, fil.nchans);
    throw std::invalid_argument(error_msg);
  }
  if (time_downsample < 1) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid time_downsample value %d, "
             "must be greater than 1",
             time_downsample);
    throw std::invalid_argument(error_msg);
  }
  if (dm_low > dm_high || dm_low < 0 || dm_step <= 0) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid DM range [%.3f-%.3f] with step %.3f", dm_low, dm_high,
             dm_step);
  }
  if (t_sample > fil.ndata * fil.tsamp) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid t_sample value %.3f, must be less than %.3f", t_sample,
             fil.ndata * fil.tsamp);
  }

  const size_t nchans = fil.nchans;
  const size_t dm_steps = static_cast<size_t>((dm_high - dm_low) / dm_step) + 1;

  const float ref_freq_value = ref_freq ? fil.frequency_table[chan_end]
                                        : fil.frequency_table[chan_start];

  int *d_delay_table;
  CHECK_CUDA(hipMallocManaged(
      &d_delay_table, dm_steps * (chan_end - chan_start + 1) * sizeof(int)));

  double *d_freq_table;
  CHECK_CUDA(hipMallocManaged(&d_freq_table, nchans * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_freq_table, fil.frequency_table,
                        nchans * sizeof(double), hipMemcpyHostToDevice));

  dim3 block_size(64, 16);
  dim3 grid_size((dm_steps + block_size.x - 1) / block_size.x,
                 (chan_end - chan_start + 1 + block_size.y - 1) / block_size.y);

  pre_calculate_dedispersion_kernel<<<grid_size, block_size>>>(
      d_delay_table, dm_low, dm_high, dm_step, chan_start, chan_end,
      d_freq_table, ref_freq_value, fil.tsamp);

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
  // check the delay table
  int *delay_table = new int[dm_steps * (chan_end - chan_start + 1)];
  CHECK_CUDA(hipMemcpy(delay_table, d_delay_table,
                        dm_steps * (chan_end - chan_start + 1) * sizeof(int),
                        hipMemcpyDeviceToHost));
  // PRINT_ARR(delay_table, dm_steps * (chan_end - chan_start + 1));

  const size_t samples_per_tsample =
      static_cast<size_t>(std::round(t_sample / fil.tsamp));
  const size_t total_slices =
      (fil.ndata + samples_per_tsample - 1) / samples_per_tsample;

  const size_t down_ndata_t =
      (samples_per_tsample + time_downsample - 1) / time_downsample;

  T *d_input;
  uint64_t *d_output;
  T *data = static_cast<T *>(fil.data);
  CHECK_CUDA(hipMalloc(&d_input, fil.ndata * nchans * sizeof(T)));
  CHECK_CUDA(hipMalloc(&d_output, dm_steps * down_ndata_t * sizeof(uint64_t)));
  CHECK_CUDA(
      hipMemset(d_output, 0, dm_steps * down_ndata_t * sizeof(uint64_t)));

  CHECK_CUDA(hipMemcpy(d_input, data, fil.ndata * nchans * sizeof(T),
                        hipMemcpyHostToDevice));

  dedisperseddata result;

  std::vector<std::shared_ptr<uint64_t[]>> dm_times;
  float total_time = fil.ndata * fil.tsamp;
  printf("Total time: %.3f s\n", total_time);
  for (size_t slice_idx = 0; slice_idx < total_slices - 1; ++slice_idx) {
    const size_t start = slice_idx * samples_per_tsample;
    const size_t end =
        std::min(start + samples_per_tsample, static_cast<size_t>(fil.ndata));
    const size_t slice_duration = end - start;
    const size_t down_ndata =
        (slice_duration + time_downsample - 1) / time_downsample;
    // printf("ndata: %zu\n", fil.ndata);
    // printf("totolidx: %zu\n", fil.ndata * fil.nchans);
    // printf("current times: %.3f s\n", start * fil.tsamp);
    // printf("current slice_idx: %zu\n", slice_idx);
    // printf("end_time: %.3f s\n", end * fil.tsamp);
    // printf("current_idata: %zu\n", start * fil.nchans);
    // printf("current_slice_duration: %zu\n", slice_duration);
    // printf("end_idata: %zu\n", end * fil.nchans);

    if (slice_idx == 0) {
      result.downtsample_ndata = down_ndata;
      result.shape = {dm_steps, down_ndata};
      // PRINT_VAR(down_ndata);
      // PRINT_VAR(result.shape[0]);
      // PRINT_VAR(result.shape[1]);
    }
    CHECK_CUDA(
        hipMemset(d_output, 0, dm_steps * down_ndata_t * sizeof(uint64_t)));

    int THREADS_PER_BLOCK = 256;
    dim3 threads(THREADS_PER_BLOCK);
    dim3 grids((down_ndata + threads.x - 1) / threads.x, dm_steps);

    dedispersion_kernel<T><<<grids, threads>>>(
        d_output, d_input, d_delay_table, dm_steps, time_downsample, fil.ndata,
        nchans, chan_start, chan_end, start, down_ndata);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    auto dm_array = std::shared_ptr<uint64_t[]>(
        new (std::align_val_t{4096}) uint64_t[dm_steps * down_ndata_t](),
        [](uint64_t *p) { operator delete[](p, std::align_val_t{4096}); });

    CHECK_CUDA(hipMemcpy(dm_array.get(), d_output,
                          dm_steps * down_ndata * sizeof(uint64_t),
                          hipMemcpyDeviceToHost));

    dm_times.emplace_back(std::move(dm_array));
  }
  CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));

  result.dm_times = std::move(dm_times);
  result.dm_low = dm_low;
  result.dm_high = dm_high;
  result.dm_step = dm_step;
  result.tsample = t_sample;
  result.filname = fil.filename;
  result.dm_ndata = dm_steps;

  return result;
}

template dedisperseddata
dedispered_fil_cuda<uint8_t>(Filterbank &fil, float dm_low, float dm_high,
                             float freq_start, float freq_end, float dm_step,
                             int ref_freq, int time_downsample, float t_sample);

template dedisperseddata
dedispered_fil_cuda<uint16_t>(Filterbank &fil, float dm_low, float dm_high,
                              float freq_start, float freq_end, float dm_step,
                              int ref_freq, int time_downsample,
                              float t_sample);

template dedisperseddata
dedispered_fil_cuda<uint32_t>(Filterbank &fil, float dm_low, float dm_high,
                              float freq_start, float freq_end, float dm_step,
                              int ref_freq, int time_downsample,
                              float t_sample);

template <typename T>
dedisperseddata dedisperse_spec(T *data, Header header, float dm_low,
                                float dm_high, float freq_start, float freq_end,
                                float dm_step, int ref_freq,
                                int time_downsample, float t_sample) {
  // Calculate frequency table from header info
  const size_t nchans = header.nchans;
  std::vector<double> frequency_table(nchans);
  float fch1 = header.fch1;
  float foff = header.foff;

  for (size_t i = 0; i < nchans; ++i) {
    frequency_table[i] = fch1 + i * foff;
  }

  float freq_min = frequency_table[0];
  float freq_max = frequency_table[nchans - 1];

  // Validate parameters
  if (freq_start < freq_min || freq_end > freq_max) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Frequency range [%.3f-%.3f MHz] out of spectrum range "
             "[%.3f-%.3f MHz]",
             freq_start, freq_end, freq_min, freq_max);
    throw std::invalid_argument(error_msg);
  }

  size_t chan_start = static_cast<size_t>((freq_start - freq_min) /
                                          (freq_max - freq_min) * (nchans - 1));
  size_t chan_end = static_cast<size_t>((freq_end - freq_min) /
                                        (freq_max - freq_min) * (nchans - 1));

  chan_start = std::max(static_cast<size_t>(0), chan_start);
  chan_end = std::min(static_cast<size_t>(nchans) - 1, chan_end);

  if (chan_start >= nchans || chan_end >= nchans) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid channel range [%zu-%zu] for %zu channels", chan_start,
             chan_end, nchans);
    throw std::invalid_argument(error_msg);
  }

  if (time_downsample < 1) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid time_downsample value %d, "
             "must be greater than or equal to 1",
             time_downsample);
    throw std::invalid_argument(error_msg);
  }

  if (dm_low > dm_high || dm_low < 0 || dm_step <= 0) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid DM range [%.3f-%.3f] with step %.3f", dm_low, dm_high,
             dm_step);
    throw std::invalid_argument(error_msg);
  }

  if (t_sample > header.ndata * header.tsamp) {
    char error_msg[256];
    snprintf(error_msg, sizeof(error_msg),
             "Invalid t_sample value %.3f, must be less than %.3f", t_sample,
             header.ndata * header.tsamp);
    throw std::invalid_argument(error_msg);
  }

  const size_t dm_steps = static_cast<size_t>((dm_high - dm_low) / dm_step) + 1;
  const float ref_freq_value =
      ref_freq ? frequency_table[chan_end] : frequency_table[chan_start];

  // Allocate and initialize delay table on GPU
  int *d_delay_table;
  CHECK_CUDA(hipMallocManaged(
      &d_delay_table, dm_steps * (chan_end - chan_start + 1) * sizeof(int)));

  double *d_freq_table;
  CHECK_CUDA(hipMallocManaged(&d_freq_table, nchans * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_freq_table, frequency_table.data(),
                        nchans * sizeof(double), hipMemcpyHostToDevice));

  // Calculate dedispersion delay table
  dim3 block_size(64, 16);
  dim3 grid_size((dm_steps + block_size.x - 1) / block_size.x,
                 (chan_end - chan_start + 1 + block_size.y - 1) / block_size.y);

  pre_calculate_dedispersion_kernel<<<grid_size, block_size>>>(
      d_delay_table, dm_low, dm_high, dm_step, chan_start, chan_end,
      d_freq_table, ref_freq_value, header.tsamp);

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  // Process data in slices
  const size_t samples_per_tsample =
      static_cast<size_t>(std::round(t_sample / header.tsamp));
  const size_t total_slices =
      (header.ndata + samples_per_tsample - 1) / samples_per_tsample;

  const size_t down_ndata_t =
      (samples_per_tsample + time_downsample - 1) / time_downsample;

  T *d_input;
  uint64_t *d_output;
  PRINT_VAR(header.ndata * nchans * sizeof(T));
  PRINT_VAR(header.ndata);
  PRINT_VAR(nchans);
  PRINT_VAR(sizeof(T));
  CHECK_CUDA(hipMalloc(&d_input, header.ndata * nchans * sizeof(T)));
  CHECK_CUDA(hipMalloc(&d_output, dm_steps * down_ndata_t * sizeof(uint64_t)));
  CHECK_CUDA(
      hipMemset(d_output, 0, dm_steps * down_ndata_t * sizeof(uint64_t)));

  CHECK_CUDA(hipMemcpy(d_input, data, header.ndata * nchans * sizeof(T),
                        hipMemcpyHostToDevice));

  dedisperseddata result;
  std::vector<std::shared_ptr<uint64_t[]>> dm_times;

  for (size_t slice_idx = 0; slice_idx < total_slices - 1; ++slice_idx) {
    const size_t start = slice_idx * samples_per_tsample;
    const size_t end = std::min(start + samples_per_tsample,
                                static_cast<size_t>(header.ndata));
    const size_t slice_duration = end - start;
    const size_t down_ndata =
        (slice_duration + time_downsample - 1) / time_downsample;

    if (slice_idx == 0) {
      result.downtsample_ndata = down_ndata;
      result.shape = {dm_steps, down_ndata};
    }

    CHECK_CUDA(
        hipMemset(d_output, 0, dm_steps * down_ndata_t * sizeof(uint64_t)));

    int THREADS_PER_BLOCK = 256;
    dim3 threads(THREADS_PER_BLOCK);
    dim3 grids((down_ndata + threads.x - 1) / threads.x, dm_steps);

    dedispersion_kernel<T><<<grids, threads>>>(
        d_output, d_input, d_delay_table, dm_steps, time_downsample,
        header.ndata, nchans, chan_start, chan_end, start, down_ndata);
    PRINT_VAR(slice_idx);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    auto dm_array = std::shared_ptr<uint64_t[]>(
        new (std::align_val_t{4096}) uint64_t[dm_steps * down_ndata_t](),
        [](uint64_t *p) { operator delete[](p, std::align_val_t{4096}); });

    CHECK_CUDA(hipMemcpy(dm_array.get(), d_output,
                          dm_steps * down_ndata * sizeof(uint64_t),
                          hipMemcpyDeviceToHost));

    dm_times.emplace_back(std::move(dm_array));
  }

  // Clean up GPU resources
  CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));
  CHECK_CUDA(hipFree(d_delay_table));
  CHECK_CUDA(hipFree(d_freq_table));

  // Fill result structure
  result.dm_times = std::move(dm_times);
  result.dm_low = dm_low;
  result.dm_high = dm_high;
  result.dm_step = dm_step;
  result.tsample = t_sample;
  result.filname = header.filename;
  result.dm_ndata = dm_steps;

  return result;
}

template dedisperseddata
dedisperse_spec<uint8_t>(uint8_t *data, Header header, float dm_low,
                         float dm_high, float freq_start, float freq_end,
                         float dm_step, int ref_freq, int time_downsample,
                         float t_sample);

template dedisperseddata
dedisperse_spec<uint16_t>(uint16_t *data, Header header, float dm_low,
                          float dm_high, float freq_start, float freq_end,
                          float dm_step, int ref_freq, int time_downsample,
                          float t_sample);
template dedisperseddata
dedisperse_spec<uint32_t>(uint32_t *data, Header header, float dm_low,
                          float dm_high, float freq_start, float freq_end,
                          float dm_step, int ref_freq, int time_downsample,
                          float t_sample);

} // namespace gpucal
